#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include "lenia.h"

__device__ float growth_function(float input, float growth_center, float growth_width) {
    return 2 * exp(-pow(input - growth_center, 2) / (2 * pow(growth_width, 2))) - 1; // Gaussian function [0, 1] -> [-1, 1]
}

__device__ float convolution(int centerx, int centery, float* frame, int frame_width, int frame_height, float* kernel, int kernel_radius) {
    float value = 0.0f;
    int kernel_size = kernel_radius * 2 + 1;

    for (int conv_x = 0; conv_x < kernel_size; ++conv_x) {
        for (int conv_y = 0; conv_y < kernel_size; ++conv_y) {
            int imagex = (centerx + (conv_x - kernel_radius));
            int imagey = (centery + (conv_y - kernel_radius));

            // % (modulo) does not work -_- (figured it out after 5 hours...)
            // if (imagex < 0) imagex += frame_width;
            // if (imagex >= frame_width) imagex -= frame_width;
            // if (imagey < 0) imagey += frame_height;
            // if (imagey >= frame_width) imagey -= frame_height;
            
            if (imagex < 0 || imagex >= frame_width || imagey < 0 || imagey >= frame_height) {
                continue;
            }

            value += frame[imagex + imagey * frame_width] * kernel[conv_x + conv_y * kernel_size];
        }
    }

    return value;
}

__global__ void cudaLenia(float *frames, float* lastFrame, int frameWidth, int frameHeight, int frameIndex, float* kernel, int kernalRadius, float growthCenter, float growthWidth, float deltaTime, int blockx, int blocky, bool saveFrame) {
    for (int imagex = blockIdx.x * blockDim.x + threadIdx.x; imagex < frameWidth; imagex += blockDim.x * blockx) {
        for (int imagey = blockIdx.y * blockDim.y + threadIdx.y; imagey < frameHeight; imagey += blockDim.y * blocky) {
            if (imagex >= frameWidth || imagey >= frameHeight)
                return;

            float value = lastFrame[imagex + imagey * frameWidth];
            float convolutionValue = convolution(imagex, imagey, &frames[frameWidth * frameHeight * (frameIndex - 1)], frameWidth, frameHeight, kernel, kernalRadius);
            float growthValue = growth_function(convolutionValue, growthCenter, growthWidth);
            float newValue = value + growthValue * deltaTime;
            newValue = fmaxf(0, fminf(newValue, 1));
            lastFrame[imagex + imagey * frameWidth] = newValue;
            if (saveFrame) frames[(imagex + imagey * frameWidth) + (frameHeight * frameWidth * frameIndex)] = newValue;
        }
    }
}

LeniaResult leniaRunCuda(LeniaData data, std::vector<float> input, KernelData kernel) {
    float* frames_cu;
    float* lastFrame_cu;
    float* kernal_cu;

	hipMalloc((void**)&frames_cu, data.FrameWidth * data.FrameHeight * sizeof(float) * data.FrameAmount);
	hipMalloc((void**)&lastFrame_cu, data.FrameWidth * data.FrameHeight * sizeof(float));
	hipMalloc((void**)&kernal_cu, kernel.Kernel.size() * sizeof(float));

    hipMemcpy(frames_cu, input.data(), data.FrameWidth * data.FrameHeight * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(lastFrame_cu, input.data(), data.FrameWidth * data.FrameHeight * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(kernal_cu, kernel.Kernel.data(), kernel.Kernel.size() * sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t startEvt, stopEvt; 
	hipEventCreate(&startEvt);
	hipEventCreate(&stopEvt);

	hipEventRecord(startEvt);
    for (int frameIndex = 1; frameIndex < data.FrameAmount; ++frameIndex) {
        for (int i = 0; i < data.IterationPerFrame; ++i) {
            cudaLenia<<<dim3(data.Blocks_x, data.Blocks_y), dim3(data.Threads_x, data.Threads_y)>>>(
                frames_cu, lastFrame_cu, 
                data.FrameWidth, data.FrameHeight, 
                frameIndex, 
                kernal_cu, kernel.Radius, 
                data.GrowthCenter, data.GrowthWidth, data.DeltaTime,
                data.Blocks_x, data.Blocks_y,
                i == data.IterationPerFrame - 1
            );
        }
    }
	hipError_t error = hipGetLastError();
	hipEventRecord(stopEvt);
	
    std::vector<float> output(data.FrameWidth * data.FrameHeight * data.FrameAmount);
	hipMemcpy(output.data(), frames_cu, data.FrameWidth * data.FrameHeight * sizeof(float) * data.FrameAmount, hipMemcpyDeviceToHost);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, startEvt, stopEvt);

	hipEventDestroy(startEvt);
	hipEventDestroy(stopEvt);
    hipFree(frames_cu);
    hipFree(lastFrame_cu);
    hipFree(kernal_cu);

    return LeniaResult(elapsedTime, hipGetErrorString(error), output);
}